/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <errno.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/ioctl.h>
#include <sys/stat.h>
#include <sys/types.h>
#include "../kernel-module/picoevb-rdma-ioctl.h"

#define MAX_TRANSFER_SIZE (100 * 1024 * 1024)

int main(int argc, char **argv)
{
	int fd, ret;
	struct picoevb_rdma_card_info card_info;
	uint64_t transfer_size;
	hipError_t ce;
	void *buf;
	hipError_t cr;
	unsigned int flag = 1;
	struct picoevb_rdma_pin_cuda pin_params;
	struct picoevb_rdma_h2c_dma dma_params;
	uint64_t tdelta_us;
	struct picoevb_rdma_unpin_cuda unpin_params;

	if (argc != 1) {
		fprintf(stderr, "usage: rdma-cuda-h2c-perf\n");
		return 1;
	}

	fd = open("/dev/picoevb", O_RDWR);
	if (fd < 0) {
		perror("open() failed");
		return 1;
	}

	ret = ioctl(fd, PICOEVB_IOC_CARD_INFO, &card_info);
	if (ret != 0) {
		fprintf(stderr, "ioctl(CARD_INFO) failed: %d\n", ret);
		perror("ioctl() failed");
		return 1;
	}
	transfer_size = card_info.fpga_ram_size;
	if (transfer_size > MAX_TRANSFER_SIZE)
		transfer_size = MAX_TRANSFER_SIZE;

#ifdef NV_BUILD_DGPU
	ce = hipMalloc(&buf, transfer_size);
#else
	ce = hipHostAlloc(&buf, transfer_size, hipHostMallocDefault);
#endif
	if (ce != hipSuccess) {
		fprintf(stderr, "Allocation of GPU buffer failed: %d\n", ce);
		return 1;
	}

	cr = hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS,
		(hipDeviceptr_t)buf);
	if (cr != hipSuccess) {
		fprintf(stderr, "hipPointerSetAttribute(buf) failed: %d\n", cr);
		return 1;
	}

	pin_params.va = (__u64)buf;
	pin_params.size = transfer_size;
	ret = ioctl(fd, PICOEVB_IOC_PIN_CUDA, &pin_params);
	if (ret != 0) {
		fprintf(stderr, "ioctl(PIN_CUDA buf) failed: ret=%d errno=%d\n", ret, errno);
		return 1;
	}

	ce = hipDeviceSynchronize();
	if (ce != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize() failed: %d\n", ce);
		return 1;
	}

	dma_params.src = pin_params.handle;
	dma_params.dst = 0;
	dma_params.len = transfer_size;
	dma_params.flags = PICOEVB_H2C_DMA_FLAG_SRC_IS_CUDA;
	ret = ioctl(fd, PICOEVB_IOC_H2C_DMA, &dma_params);
	if (ret != 0) {
		fprintf(stderr, "ioctl(DMA) failed: %d\n", ret);
		perror("ioctl() failed");
		return 1;
	}

	tdelta_us = dma_params.dma_time_ns / 1000;
	printf("Bytes:%lu usecs:%lu MB/s:%lf\n", transfer_size, tdelta_us, (double)transfer_size / (double)tdelta_us);

	unpin_params.handle = pin_params.handle;
	ret = ioctl(fd, PICOEVB_IOC_UNPIN_CUDA, &unpin_params);
	if (ret != 0) {
		fprintf(stderr, "ioctl(UNPIN_CUDA buf) failed: %d\n", ret);
		return 1;
	}

#ifdef NV_BUILD_DGPU
	ce = hipFree(buf);
#else
	ce = hipHostFree(buf);
#endif
	if (ce != hipSuccess) {
		fprintf(stderr, "Free of buf failed: %d\n", ce);
		return 1;
	}

	ret = close(fd);
	if (ret < 0) {
		perror("close() failed");
		return 1;
	}

	return 0;
}
